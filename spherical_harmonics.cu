#include "hip/hip_runtime.h"
#include "spherical_harmonics.h"
#include <limits>
#include <omp.h>

using glm::vec2;
using glm::vec3;

std::vector<vec2> uniform_sphere_2d_samples(int n) {
    std::vector<vec2> ret(n);
    for(int i = 0;i < n; ++i) {
        float x = pd::random_float();
        float y = pd::random_float();
        ret[i] = vec2(2.0f * std::acos(std::sqrt(1.0f - x)), 2.0f * pd::pi * y);
    }
    return ret;
}

std::vector<vec3> uniform_sphere_3d_samples(int n) {
    auto samples = uniform_sphere_2d_samples(n);
    std::vector<vec3> ret(n);

    for(int i = 0; i < n; ++i) {
        float a = samples[i].x, b = samples[i].y;
        vec3 p(std::sin(a) * std::cos(b), std::cos(a),std::sin(a) * std::sin(b));
        ret[i] = p;
    }
    return ret;
}

long long factorial(int l) {
    long long ret = 1;
    for(int i = 2; i <= l; ++i) {
        ret = ret * i;
    }
    return ret;
}

long long dfactorial(int l) {
    long long ret = 1;
    for(int i = l; i >= 2.0; i=i-2) {
        ret *= i;
    }
    return ret;
}

float K(int l, int m) {
    float k = ((2.0f * l + 1.0f) * factorial(l-std::abs(m)))/(4.0f * pd::pi * factorial(l + std::abs(m)));
    return std::sqrt(k);
}

float P(int l, int m, float x) {
    if (m == 0 && l == 0) {
        return 1.0f;
    }

    if (l == m) {
        float sqrt_t = std::sqrt(1.0-x * x);
        return std::pow(-1, m) * dfactorial(2 * m - 1) * std::pow(sqrt_t, m);
    }
    
    if (l == m+1) {
        return x * (2.0f * m + 1.0f) * P(m,m,x);
    }

    return (x * (2.0f * l - 1.0f) * P(l-1, m, x) - (l+m-1) * P(l-2, m, x))/(l-m);
}

const float sqrt2 = std::sqrt(2.0f);
float SH(int l, int m, float theta, float phi) {
    if (m==0) 
        return K(l,0) * P(l, 0, cos(theta));
    
    if (m > 0) 
        return sqrt2 * K(l,m) * cos(m * phi) * P(l, m, cos(theta));
    
    return sqrt2 * K(l,-m) * sin(-m * phi) * P(l, -m, cos(theta)); 
}

std::vector<SH_sample> SH_init(int band, int num) {
    int sample_num = num * band * band;
    
    // memory layout: 
    // band, series x N samples 
    std::vector<SH_sample> ret(sample_num);

    auto samples = uniform_sphere_2d_samples(num);
    int sample_i = 0;
    for(int l = 0; l < band; ++l) {
        for(int m = -l; m <= l; ++m) {
            for (int i = 0; i < num; ++i) {
                float a = samples[i].x, b = samples[i].y;
                vec3 p(std::sin(a) * std::cos(b), std::cos(a),std::sin(a) * std::sin(b));
                
                ret[sample_i].sph = samples[i];
                ret[sample_i].vec = p;

                // coefficients
                ret[sample_i].c = SH(l, m, ret[sample_i].sph.x, ret[sample_i].sph.y);
                sample_i += 1;
            }
        }
    }

    return ret;
}

// return a sparse band/series matrix
std::vector<float> SH_func(std::function<float(float theta, float phi)> func, const std::vector<SH_sample> &samples, int band) {
    int band_n = band * band;
    std::vector<float> ret(band_n);
    int sample_num = (int)samples.size() / band_n;
    int ind = 0;

    const float mc_factor = 4.0f * pd::pi /(float)sample_num;
    for (int l = 0; l < band; ++l) {
        for(int m = -l; m <=l; ++m) {
            float c = 0.0f;

            // monte-carlo integration
            for(int si = 0; si < sample_num; ++si) {
                c += func(samples[si].sph.x, samples[si].sph.y) * samples[si].c * mc_factor;
            }

            ret[ind++] = c;
        }
    }

    return ret; 
}

void compute_sh_coeff(std::shared_ptr<mesh> mesh_ptr, int band, int n) {
	if(!mesh_ptr) return;

	if(mesh_ptr->m_norms.empty()) {
		mesh_ptr->recompute_normal();
	}

	mesh_ptr->m_band = band;
	mesh_ptr->m_sh_coeffs.resize(band * band * mesh_ptr->m_norms.size());
	
    auto sh_samples = SH_init(band, n);
	int sample_num = (int)sh_samples.size()/(band * band);
	const float mc_factor = 4.0f * pd::pi / (float)sample_num;

#pragma omp parallel for
	for(int vi = 0; vi < mesh_ptr->m_norms.size(); ++vi) {
        int ind = 0;
		for(int l = 0; l < band; ++l) for(int m = -l; m <=l; ++m) {
			float c = 0.0f;
			// monte-carlo integration
			for(int si = 0; si < sample_num; ++si) {
				c +=
				std::max(glm::dot(mesh_ptr->m_norms[vi], sh_samples[ind * sample_num + si].vec), 0.0f) * 
				sh_samples[ind * sample_num + si].c * 
				mc_factor;
			}
			mesh_ptr->m_sh_coeffs[vi * band * band + ind++] = c;		
		}
	}
	
	mesh_ptr->compute_sh_texture();
}

__global__
void cuda_no_shadow(glm::vec3 *norms, int norm_n, SH_sample *sh_samples, int sh_n, int band, float *d_coeffs) {
    int ind = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    int sample_num = sh_n / (band * band);
    float mc_factor = 4.0f * 3.1415926f/(float)sample_num;
    for(int vi = ind; vi < norm_n; vi += stride) {
        int ind = 0;
        for(int l = 0; l < band; ++l) for(int m = -l; m <= l; ++m) {
            float c = 0.0f;
            for(int si = 0; si < sample_num; ++si) {
                float dot_term = glm::dot(norms[vi], sh_samples[ind * sample_num + si].vec);
                if (dot_term < 0.0) dot_term = 0.0f;

                c += dot_term * sh_samples[ind * sample_num + si].c * mc_factor;
            }
            d_coeffs[vi * band * band + ind++] = c;
        }
    }
}

void cuda_compute_sh_coeff(std::shared_ptr<mesh> mesh_ptr, int band, int n) {
	if(!mesh_ptr) return;

	if(mesh_ptr->m_norms.empty()) {
		mesh_ptr->recompute_normal();
	}

	mesh_ptr->m_band = band;
	mesh_ptr->m_sh_coeffs.resize(band * band * mesh_ptr->m_norms.size());
	
    auto sh_samples = SH_init(band, n);
	int sample_num = (int)sh_samples.size()/(band * band);

    // memory allocation
    glm::vec3 *d_norms;         size_t d_norms_size = sizeof(glm::vec3) * mesh_ptr->m_norms.size();
    SH_sample *d_sh_samples;    size_t d_sh_samples_size = sizeof(SH_sample) * sh_samples.size();
    float *d_coeffs;            size_t d_coeffs_size = sizeof(float) * band * band * mesh_ptr->m_norms.size();

    GC(hipMalloc(&d_norms, d_norms_size));
    GC(hipMalloc(&d_sh_samples, d_sh_samples_size));
    GC(hipMalloc(&d_coeffs, d_coeffs_size));
    GC(hipMemcpy(d_norms, mesh_ptr->m_norms.data(), d_norms_size, hipMemcpyHostToDevice));
    GC(hipMemcpy(d_sh_samples, sh_samples.data(), d_norms_size, hipMemcpyHostToDevice))

    // cuda computation
    int grid = 512, block = (grid + mesh_ptr->m_norms.size() -1)/grid;
    cuda_no_shadow<<<grid,block>>>(d_norms, mesh_ptr->m_norms.size(), d_sh_samples, sh_samples.size(), band, d_coeffs);
    GC(hipDeviceSynchronize());

    // memory copy back
    GC(hipMemcpy(mesh_ptr->m_sh_coeffs.data(),d_coeffs, d_coeffs_size, hipMemcpyDeviceToHost));

    // memory free
    hipFree(d_norms);
    hipFree(d_sh_samples);

    mesh_ptr->compute_sh_texture();
}